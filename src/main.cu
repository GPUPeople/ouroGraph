#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <map>
#include <fstream>
#include <iomanip>
#include <ctime>
#include <sstream>

#include "CSR.h"
#include "dCSR.h"
#include "COO.h"
#include "device/ouroGraph_impl.cuh"
#include "device/Initialization.cuh"
#include "InstanceDefinitions.cuh"
#include "MemoryLayout.h"
#include "Verification.h"

// Json Reader
#include "helper/json.h"

// Using declarations
using json = nlohmann::json;
using DataType = float;

int main(int argc, char* argv[])
{
	if (argc == 1)
	{
		std::cout << "Require config file as first argument" << std::endl;
		return -1;
	}
	printf("%souroGraph - Test Application\n%s", break_line_blue_s, break_line_blue_e);

	// Parse config
	std::ifstream json_input(argv[1]);
	json config;
	json_input >> config;

	// Device configuration
	const auto device{config.find("device").value().get<int>()};
	hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << "\n";

	auto graphs = *config.find("graphs");
	for(auto const& elem : graphs)
	{
		std::string filename = elem.find("filename").value().get<std::string>();
		CSR<DataType> csr_graph;
		//try load csr file
		std::string csr_name = filename + ".csr";
		printTestcaseSeparator(filename);
		try
		{
			std::cout << "trying to load csr file \"" << csr_name << "\"\n";
			csr_graph = loadCSR<DataType>(csr_name.c_str());
			std::cout << "succesfully loaded: \"" << csr_name << "\"\n";
		}
		catch (std::exception& ex)
		{
			std::cout << "could not load csr file:\n\t" << ex.what() << "\n";
			try
			{
				filename += std::string(".mtx");
				std::cout << "trying to load mtx file \"" << filename << "\"\n";
				auto coo_mat = loadMTX<DataType>(filename.c_str());
				convert(csr_graph, coo_mat);
				std::cout << "succesfully loaded and converted: \"" << csr_name << "\"\n";
			}
			catch (std::exception& ex)
			{
				std::cout << ex.what() << std::endl;
				return -1;
			}
			try
			{
				std::cout << "write csr file for future use\n";
				storeCSR(csr_graph, csr_name.c_str());
			}
			catch (std::exception& ex)
			{
				std::cout << ex.what() << std::endl;
			}
		}

		std::cout << "Using: " << argv[1] << " with " << csr_graph.rows << " vertices and " << csr_graph.nnz << " edges\n";
		if(printDebug)
		{
			auto max_adjacency_length = 0U;
			auto min_adjacency_length = 0xFFFFFFFFU;
			for(auto i = 0U; i < csr_graph.rows; ++i)
			{
				auto neighbours = csr_graph.row_offsets[i + 1] - csr_graph.row_offsets[i];
				max_adjacency_length = std::max(max_adjacency_length, neighbours);
				min_adjacency_length = std::min(min_adjacency_length, neighbours);
			}
			std::cout << "Smallest Adjacency: " << min_adjacency_length << " | Largest Adjacency: " << max_adjacency_length << " | Average Adjacency: "
			<< csr_graph.row_offsets[csr_graph.rows] / csr_graph.rows << "\n";
		}

		// FLush Graph beforehand
		for(auto i = 0; i < csr_graph.rows; ++i)
		{
			auto offset = csr_graph.row_offsets[i];
			auto neighbours = csr_graph.row_offsets[i + 1] - offset;
			for(auto j = 0; j < neighbours; ++j)
			{
				csr_graph.col_ids[offset + j] = i;
			}
		}

		// Graph Testcase
		ouroGraph<VertexData, EdgeData, OuroPQ> graph;
		graph.initialize(csr_graph);
		CSR<DataType> csr_output;
		graph.ouroGraphToCSR(csr_output);

		// Verification
		Verification<DataType> verification(csr_graph);
		verification.verify(csr_output, "Initialization", OutputCodes::VERIFY_INITIALIZATION);
	}

	return 0;
}